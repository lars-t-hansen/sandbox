#include "hip/hip_runtime.h"
/* -*- fill-column: 100 -*- */
/* Cuda mandelbrot */

#include "hip/hip_runtime.h"

/* Several interesting complications here:

   - The `iterations` array must be allocated as Cuda host memory in order to optimize the memcpy
     (otherwise the memcpy is very slow); not a free lunch though: this memory is pinned in RAM and
     we can't have an unbounded amount

   - Host malloc is also expensive and the expense grows with the size.  It is more expensive than
     the computation in our case.  So the allocation cost must be amortized.

   - The cuda initialization takes a long time and is measured separately, and in practice it means
     that we must amortize the cost of it across many runs of computation.

   - Default tile sizes are such that tile_x*tile_y==32, the size of a warp.  Empirically this
     appears pretty much to be optimal.  The warp will only exit once all threads are done, so if
     one thread gets stuck in a deep search while the others exit we're wasting time.  But it seems
     likely (fractals...)  that no particular shape of a tile will be better than another, only
     smaller tiles could be better, and then only if scheduling is free.  But my understanding is
     that no tile can effectively be smaller than a warp without basically wasting resources. */

static unsigned tile_y = 2;
static unsigned tile_x = 16;

/* TODO: Can the host memory be mapped into the cuda address space so as to avoid the memcpy?  See
   the hipHostMallocMapped flag to hipHostAlloc. The code runs but the output is borked, maybe need
   some kind of sync?  Or is the code incomplete.  There is hipHostGetDevicePointer and in the
   description of that there's wording about registering things, too. */

//#define USE_MAPPED_MEMORY_IF_POSSIBLE

/* Canvas size in pixels */
#define WIDTH 1400
#define HEIGHT 800

/* Classic mandelbrot set */
typedef float float_t;
static const unsigned CUTOFF = 3000;
static const float_t MINY = -1;
static const float_t MAXY = 1;
static const float_t MINX = -2.5;
static const float_t MAXX = 1;

static unsigned* iterations; // Cuda host memory, [HEIGHT * WIDTH]

#include "../mandelcommon/mandelcommon.h"

__device__ inline float_t scale(float_t v, float_t rng, float_t min, float_t max) {
  return min + v*((max-min)/rng);
}

__device__ unsigned mandel_pixel(unsigned py, unsigned px) {
  float_t y0 = scale(py, HEIGHT, MINY, MAXY);
  float_t x0 = scale(px, WIDTH, MINX, MAXX);
  float_t x = 0, y = 0;
  unsigned iteration = 0;
  while (x*x+y*y <= 4 && iteration < CUTOFF) {
    float_t nx = x*x - y*y + x0;
    float_t ny = 2*x*y + y0;
    x = nx;
    y = ny;
    iteration++;
  }
  return iteration;
}

__global__ void mandel_worker(unsigned* iterations) {
  unsigned row = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned col = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < HEIGHT && col < WIDTH) {
    iterations[row*WIDTH + col] = mandel_pixel(row, col);
  }
}

static bool can_map_memory = false;

static void initCuda() {
  /* Get the device ID and sync to force initialization so that it doesn't pollute timings.  Getting
     the device ID is by itself not enough. */
  begin_timer();
  int dev_id;
  hipGetDevice(&dev_id);
  unsigned flags;
  hipGetDeviceFlags(&flags);
#ifdef USE_MAPPED_MEMORY_IF_POSSIBLE
  can_map_memory = flags & hipDeviceMapHost;
#endif
  hipDeviceSynchronize();
  end_timer("init");
#ifndef NDEBUG
  fprintf(stderr, "device %d\n", dev);
#endif
}

static void initHostMemory() {
  size_t nbytes = HEIGHT*WIDTH*sizeof(unsigned);
  hipError_t err;
  begin_timer();
  if (can_map_memory) {
    if ((err = hipHostAlloc(&iterations, nbytes, hipHostMallocMapped)) != 0) {
      fprintf(stderr, "host alloc mapped %zu bytes %d\n", nbytes, err);
      abort();
    }
  } else {
    if ((err = hipHostMalloc(&iterations, nbytes)) != 0) {
      fprintf(stderr, "host malloc %zu bytes %d\n", nbytes, err);
      abort();
    }
  }
  end_timer("Host malloc");
}

static void mandel() {
  size_t nbytes = HEIGHT*WIDTH*sizeof(unsigned);
  unsigned *dev_iterations;
  hipError_t err;

  begin_timer();
  if ((err = hipMalloc(&dev_iterations, nbytes)) != 0) {
    fprintf(stderr, "device malloc %zu bytes %d\n", nbytes, err);
    abort();
  }
  end_timer("Device malloc");

  dim3 threadsPerBlock(tile_x, tile_y);
  dim3 blocksPerGrid((WIDTH+tile_x-1)/tile_x, (HEIGHT+tile_y-1)/tile_y);
  begin_timer();
  mandel_worker<<<blocksPerGrid, threadsPerBlock>>>(dev_iterations);
  hipDeviceSynchronize();
  end_timer("Compute");

  if (!can_map_memory) {
    begin_timer();
    if ((err = hipMemcpy(iterations, dev_iterations, nbytes, hipMemcpyDeviceToHost)) != 0) {
      fprintf(stderr, "memcpy %d\n", err);
      abort();
    }
    hipDeviceSynchronize();
    end_timer("Memcpy");
  } else {
    // Probably need some kind of synchronization?
  }

  begin_timer();
  hipFree(dev_iterations);
  end_timer("Free");
}

int main(int argc, char** argv) {
  for (int i=1 ; i < argc; i++ ) {
    if (sscanf(argv[i], "-y%u", &tile_y) == 1) {
      if (tile_y == 0) {
	fprintf(stderr, "Zero rows\n");
	exit(1);
      }
      continue;
    }
    if (sscanf(argv[i], "-x%u", &tile_x) == 1) {
      if (tile_x == 0) {
	fprintf(stderr, "Zero columns\n");
	exit(1);
      }
      continue;
    }
    fprintf(stderr, "Bad option %s\n", argv[1]);
  }

  initCuda();
  initHostMemory();
  mandel();
  dump("mandelcuda.ppm");
}
