/* Cuda mandelbrot */

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <inttypes.h>
#include <assert.h>
#include "hip/hip_runtime.h"

/* Canvas size in pixels */
#define WIDTH 1400
#define HEIGHT 800

/* Classic mandelbrot set */
typedef float float_t;
static const unsigned CUTOFF = 3000;
static const float_t MINY = -1;
static const float_t MAXY = 1;
static const float_t MINX = -2.5;
static const float_t MAXX = 1;

#define RGB(r, g, b) ((r << 16) | (g << 8) | b)

/* Supposedly the gradients used by the Wikipedia mandelbrot page */
static unsigned mapping[] = {
  RGB(66, 30, 15),
  RGB(25, 7, 26),
  RGB(9, 1, 47),
  RGB(4, 4, 73),
  RGB(0, 7, 100),
  RGB(12, 44, 138),
  RGB(24, 82, 177),
  RGB(57, 125, 209),
  RGB(134, 181, 229),
  RGB(211, 236, 248),
  RGB(241, 233, 191),
  RGB(248, 201, 95),
  RGB(255, 170, 0),
  RGB(204, 128, 0),
  RGB(153, 87, 0),
  RGB(106, 52, 3),
};

static unsigned iterations[HEIGHT * WIDTH];

static void from_rgb(unsigned rgb, unsigned* r, unsigned* g, unsigned* b) {
  *r = (rgb >> 16) & 255;
  *g = (rgb >> 8) & 255;
  *b = rgb & 255;
}

__device__ inline float_t scale(float_t v, float_t rng, float_t min, float_t max) {
  return min + v*(max-min)/rng;
}

__device__ unsigned mandel_pixel(unsigned py, unsigned px) {
  /* TODO: Overhead.  We can hoist a bunch of stuff here I think. */
  float_t y0 = scale(py, HEIGHT, MINY, MAXY);
  float_t x0 = scale(px, WIDTH, MINX, MAXX);
  float_t x = 0, y = 0;
  unsigned iteration = 0;
  while (x*x+y*y <= 4 && iteration < CUTOFF) {
    float_t nx = x*x - y*y + x0;
    float_t ny = 2*x*y + y0;
    x = nx;
    y = ny;
    iteration++;
  }
  return iteration;
}

__global__ void mandel_worker(unsigned* iterations) {
  unsigned row = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned col = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < HEIGHT && col < WIDTH) {
    iterations[row*WIDTH + col] = mandel_pixel(row, col);
  }
}

static void mandel() {
#ifndef NDEBUG
  int dev = -87;
  hipGetDevice(&dev);
  fprintf(stderr, "device %d\n", dev);
#endif

  size_t nbytes = sizeof(unsigned)*HEIGHT*WIDTH;
  assert(nbytes == sizeof(iterations));
  unsigned *dev_iterations;
  hipError_t err;
  if ((err = hipMalloc(&dev_iterations, nbytes)) != 0) {
    fprintf(stderr, "malloc %u bytes %d\n", (unsigned)nbytes, err);
    abort();
  }

  const unsigned TILEY = 4;
  const unsigned TILEX = 4;
  dim3 threadsPerBlock(TILEX, TILEY);
  dim3 blocksPerGrid((WIDTH+TILEX-1)/TILEX, (HEIGHT+TILEY-1)/TILEY);
  mandel_worker<<<blocksPerGrid, threadsPerBlock>>>(dev_iterations);

#ifndef NDEBUG
  for ( int y=0 ; y < HEIGHT; y++ ) {
    for ( int x=0 ; x < WIDTH; x++ ) {
      iterations[y*WIDTH + x] = 2;
    }
  }
#endif
  if ((err = hipMemcpy(iterations, dev_iterations, nbytes, hipMemcpyDeviceToHost)) != 0) {
    fprintf(stderr, "memcpy %d\n", err);
    abort();
  }
  hipFree(dev_iterations);
}

static void dump(const char* filename) {
  FILE* out = fopen(filename, "w");
  fprintf(out, "P6 %d %d 255\n", WIDTH, HEIGHT);
  unsigned y, x;
  for (y=0; y < HEIGHT; y++) {
    for ( x = 0 ; x < WIDTH; x++ ) {
      unsigned r = 0, g = 0, b = 0;
      if (iterations[y*WIDTH + x] < CUTOFF) {
	from_rgb(mapping[iterations[y*WIDTH + x] % 16], &r, &g, &b);
      }
      fputc(r, out);
      fputc(g, out);
      fputc(b, out);
    }
  }
  fclose(out);
}

int main(int argc, char** argv) {
  struct timeval before, after;
  gettimeofday(&before, NULL);
  mandel();
  gettimeofday(&after, NULL);
  int64_t delta = ((int64_t)after.tv_sec - (int64_t)before.tv_sec)*1000000 + (after.tv_usec - before.tv_usec);
  printf("Elapsed %" PRIi64 "ms\n", delta/1000);
  dump("mandelcuda.ppm");
}
