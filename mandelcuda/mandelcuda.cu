#include "hip/hip_runtime.h"
/* -*- fill-column: 100 -*- */
/* Cuda mandelbrot */

#include "hip/hip_runtime.h"

/* Default values are such that tile_x*tile_y==32, the size of a warp.  The warp will only exit once
   all threads are done, so if one thread gets stuck in a deep search while the others exit we're
   wasting time. */

static unsigned tile_y = 2;
static unsigned tile_x = 16;

/* Canvas size in pixels */
#define WIDTH 1400
#define HEIGHT 800

/* Classic mandelbrot set */
typedef float float_t;
static const unsigned CUTOFF = 3000;
static const float_t MINY = -1;
static const float_t MAXY = 1;
static const float_t MINX = -2.5;
static const float_t MAXX = 1;

static unsigned iterations[HEIGHT * WIDTH];

#include "../mandelcommon/mandelcommon.h"

__device__ inline float_t scale(float_t v, float_t rng, float_t min, float_t max) {
  return min + v*(max-min)/rng;
}

__device__ unsigned mandel_pixel(unsigned py, unsigned px) {
  /* TODO: Overhead.  We can hoist a bunch of stuff here I think. */
  float_t y0 = scale(py, HEIGHT, MINY, MAXY);
  float_t x0 = scale(px, WIDTH, MINX, MAXX);
  float_t x = 0, y = 0;
  unsigned iteration = 0;
  while (x*x+y*y <= 4 && iteration < CUTOFF) {
    float_t nx = x*x - y*y + x0;
    float_t ny = 2*x*y + y0;
    x = nx;
    y = ny;
    iteration++;
  }
  return iteration;
}

__global__ void mandel_worker(unsigned* iterations) {
  unsigned row = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned col = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < HEIGHT && col < WIDTH) {
    iterations[row*WIDTH + col] = mandel_pixel(row, col);
  }
}

static void mandel() {
#ifndef NDEBUG
  int dev = -87;
  hipGetDevice(&dev);
  fprintf(stderr, "device %d\n", dev);
#endif

  size_t nbytes = sizeof(iterations);
  unsigned *dev_iterations;
  hipError_t err;

  /* Just sync to force initialization so that it doesn't pollute timings */
  begin_timer();
  hipDeviceSynchronize();
  end_timer("init");

  begin_timer();
  if ((err = hipMalloc(&dev_iterations, nbytes)) != 0) {
    fprintf(stderr, "malloc %zu bytes %d\n", nbytes, err);
    abort();
  }
  end_timer("Malloc");

  dim3 threadsPerBlock(tile_x, tile_y);
  dim3 blocksPerGrid((WIDTH+tile_x-1)/tile_x, (HEIGHT+tile_y-1)/tile_y);
  begin_timer();
  mandel_worker<<<blocksPerGrid, threadsPerBlock>>>(dev_iterations);
  hipDeviceSynchronize();
  end_timer("Compute");

  begin_timer();
  if ((err = hipMemcpy(iterations, dev_iterations, nbytes, hipMemcpyDeviceToHost)) != 0) {
    fprintf(stderr, "memcpy %d\n", err);
    abort();
  }
  end_timer("Memcpy");

  begin_timer();
  hipFree(dev_iterations);
  end_timer("Free");
}

int main(int argc, char** argv) {
  for (int i=1 ; i < argc; i++ ) {
    if (sscanf(argv[i], "-y%u", &tile_y) == 1) {
      if (tile_y == 0) {
	fprintf(stderr, "Zero rows\n");
	exit(1);
      }
      continue;
    }
    if (sscanf(argv[i], "-x%u", &tile_x) == 1) {
      if (tile_x == 0) {
	fprintf(stderr, "Zero columns\n");
	exit(1);
      }
      continue;
    }
    fprintf(stderr, "Bad option %s\n", argv[1]);
  }

  mandel();
  dump("mandelcuda.ppm");
}
